#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void kernel() {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t n = tid;
	uint32_t sum = 0;
    uint32_t prod = 1;
    while(n != 0){
        uint32_t digit = n % 10;
        n /= 10;
        sum += digit;
        prod *= digit;
    }
    if(sum*prod == tid) printf("%u\n", tid);
	return;
}

void checkrange(uint32_t range){
    double dim = sqrt(range);
    printf("Checking %u for sum-product numbers\n", range);
    uint32_t blocks = (uint32_t)ceil(range/(dim));
    printf("dim: %f, blocks: %d\n", dim, blocks);
    kernel<<<(uint32_t)dim, blocks, 0>>>();
    hipError_t rc = hipDeviceSynchronize();
    printf("Return code : %d\n", rc);
    rc = hipGetLastError();
    printf("Last error : %s\n", hipGetErrorString(rc));
}

int main() {
	// main iteration
	checkrange(1024);
    checkrange(16777216);
	return 0;
}