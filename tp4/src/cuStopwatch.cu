#include <hip/hip_runtime.h>

#include <stdio.h>


class cuStopwatch{
    public:
        cuStopwatch();
        ~cuStopwatch();
        void start();
        float stop();

    private:
        float elapsedTime;
        bool started;
        hipEvent_t startTime;
        hipEvent_t endTime;
};

cuStopwatch::cuStopwatch(){
    started = false;
    elapsedTime = 0;
    hipError_t res = hipEventCreate(&startTime);
    if (res != 0)
        printf("Return code when recording startTime : %d\n", res);

    res = hipEventCreate(&endTime);
    if (res != 0)
        printf("Return code when recording endTime : %d\n", res);
}

cuStopwatch::~cuStopwatch(){
    hipEventDestroy(startTime);
    hipEventDestroy(endTime);
}

void cuStopwatch::start(){
    if (started) {
        return;
    }
    hipError_t res = hipEventRecord(startTime);
    if (res != 0)
        printf("Return code when recording startTime : %d\n", res);
    started = true;
}

float cuStopwatch::stop(){
    if (! started) {
        return 0;
    }
    hipError_t res = hipEventRecord(endTime);
    if (res != 0)
        printf("Return code when recording endTime : %d\n", res);

    hipEventSynchronize(endTime);
    
    res = hipEventElapsedTime(&elapsedTime, startTime, endTime);
    if (res != 0)
        printf("Return code when computing elapsed time : %d\n", res);
    
    started = false;
    return elapsedTime;
}

