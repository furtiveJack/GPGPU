#include <hip/hip_runtime.h>

#include <stdio.h>

// utility function provided by https://gist.github.com/jefflarkin/5390993
#define cudaCheckError() {                                          \
    hipError_t e=hipGetLastError();                                 \
    if(e!=hipSuccess) {                                              \
        printf("Cuda failure %s:%d: '%s' (err: %d)\n",__FILE__,__LINE__,hipGetErrorString(e), e);           \
        exit(0); \
    }                                                                 \
}


class cuStopwatch{
    public:
        cuStopwatch();
        ~cuStopwatch();
        void start();
        float stop();

    private:
        float elapsedTime;
        bool started;
        hipEvent_t startTime;
        hipEvent_t endTime;
};

cuStopwatch::cuStopwatch(){
    started = false;
    elapsedTime = 0;
    hipError_t res = hipEventCreate(&startTime); /*cudaCheckError();*/
    if (res != 0)
        printf("Return code when recording startTime : %d\n", res);

    res = hipEventCreate(&endTime); /*cudaCheckError();*/
    if (res != 0)
        printf("Return code when recording endTime : %d\n", res);
}

cuStopwatch::~cuStopwatch(){
    hipEventDestroy(startTime);
    hipEventDestroy(endTime);
}

void cuStopwatch::start(){
    if (started) {
        return;
    }
    hipError_t res = hipEventRecord(startTime); /*cudaCheckError();*/
    if (res != 0)
        printf("Return code when recording startTime : %d\n", res);
    started = true;
}

float cuStopwatch::stop(){
    if (! started) {
        return 0;
    }
    hipError_t res = hipEventRecord(endTime); /*cudaCheckError();*/
    if (res != 0)
        printf("Return code when recording endTime : %d\n", res);

    hipEventSynchronize(endTime); /*cudaCheckError();*/
    
    res = hipEventElapsedTime(&elapsedTime, startTime, endTime); /*cudaCheckError();*/
    if (res != 0)
        printf("Return code when computing elapsed time : %d\n", res);
    
    started = false;
    return elapsedTime;
}

