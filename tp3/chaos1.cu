#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include "cuStopwatch.cu"

// Compute sum of integers from 0 to n-1
__global__ void trianglenumber(uint64_t* res, uint64_t n) {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < n){
        // *res += tid;
        atomicAdd(res, tid); // explain the problem
    }
	return;
}

int main() {
    // Allocate memory
    uint64_t *res_host, *res_dev;
    hipHostAlloc((void**)&res_host, sizeof(uint64_t), hipHostMallocDefault);
    hipMalloc((void**)&res_dev, sizeof(uint64_t));

    hipMemset((void*)res_dev, 0, sizeof(uint64_t));

	// Perform computation
    cuStopwatch sw1;
    sw1.start();
	trianglenumber<<<1024, 1024>>>(res_dev, 1024*1024);
    hipMemcpyAsync(res_host, res_dev, sizeof(uint64_t), hipMemcpyDeviceToHost);
    printf("Computation time: %.4fms\n", sw1.stop());
    printf("Result: %I64u\n", *res_host);
    
    // Free memory
    hipFree(res_dev);
    hipHostFree(res_host);
	return 0;
}