#include "hip/hip_runtime.h"
#include "SDL_image.h"
#include <stdio.h>
#include <time.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <>
#include "cuStopwatch.cu"

enum conv_t{
    CONV_IDENTITY,
    CONV_EDGE,
    CONV_SHARP,
    CONV_GAUSS
};

SDL_Window *screen;
SDL_Window *screen_res;
SDL_Renderer *ren;
SDL_Renderer *ren_res;
SDL_Texture *tex;
SDL_Texture *tex_res;
SDL_Surface *surf;

int32_t width, height;
float filter[9];

float __constant__ filter_device[9];

__global__ void conv_global(const unsigned char* src, unsigned char* dest, int32_t w, int32_t h){
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < h * w * 3) {
        int val;
        int i = tid / w;
        int j = tid % w;
        int tid_index = tid * 3;
        int w_offset = w * 3;

        int i_min = (i == 0) ? 0 : w_offset;
        int i_max = (i == h-1) ? 0 : w_offset;
        
        int j_min = (j == 0) ? 0 : 3;
        int j_max = (j == w-1) ? 0 : 3;

        for (int k = 0 ; k < 3 ; ++k) {
            val = 0;
            val += filter_device[0] * src[k + tid_index - i_min - j_min];
            val += filter_device[1] * src[k + tid_index - i_min];
            val += filter_device[2] * src[k + tid_index - i_min + j_max];
            val += filter_device[3] * src[k + tid_index - j_min];
            val += filter_device[4] * src[k + tid_index];
            val += filter_device[5] * src[k + tid_index + j_max];
            val += filter_device[6] * src[k + tid_index + i_max - j_min];
            val += filter_device[7] * src[k + tid_index + i_max];
            val += filter_device[8] * src[k + tid_index + i_max + j_max];
            
            if (val > 255) {
                val = 255;
            } else if (val < 0) {
                val = 0;
            }
            dest[tid_index + k] = val; 
        }
    }
}

float conv_global_gpu(unsigned char* pixels, int32_t w, int32_t h){
    unsigned char* src;
    unsigned char* src_host;
    unsigned char* dest;
    float elapsed = 0;
    int size = w*h*3;
    cuStopwatch clock;

    hipHostAlloc((void**) &src_host, size*sizeof(int), hipHostMallocDefault);
    hipMalloc((void**) &src, size*sizeof(int));
    hipMalloc((void**) &dest, size*sizeof(int));
    // init src_host with pixels values
    for (int i = 0 ; i < size ; i++) {
        src_host[i] = pixels[i];
    }
    // init device with host values
    hipMemcpy(src, src_host, size*sizeof(int), hipMemcpyHostToDevice);
    // Start computation
    clock.start();
    conv_global<<<ceil(w*h / 1024), 1024>>>(src, dest, w, h);
    elapsed = clock.stop();
    // getting back the results of the computation
    hipMemcpy(src_host, dest, size*sizeof(int), hipMemcpyDeviceToHost);
    // storing computed results in pixels
    for (int i = 0 ; i < size ; ++i) {
        pixels[i] = src_host[i];
    }
    // free the memory
    hipHostFree(src_host);
    hipFree(src);
    hipFree(dest);
    return elapsed;
}

__global__ void conv_texture(hipTextureObject_t src, unsigned char* dest, int32_t w, int32_t h){
}

float conv_texture_gpu(unsigned char* pixels, int32_t w, int32_t h){
    // todo: write the code that manages memory (texture memory) and invokes the kernel conv_texture, it should return the running time
    return 0;
}

int main(int argc, char** argv){
    SDL_Event event;
    bool withtex = false;
    
    // Initialize SDL
    if( SDL_Init(SDL_INIT_VIDEO) < 0 ) {
        fprintf(stderr, "Couldn't initialize SDL: %s\n", SDL_GetError());
        exit(1);
    }
	atexit(SDL_Quit);
    
    if(argc == 1){
        exit(1);
    }
    
    // Read image and option
    IMG_Init(IMG_INIT_PNG);
    surf = IMG_Load(argv[1]);
    if(surf == NULL){
        fprintf(stderr, "Error loading image.\n");
        exit(1);
    }
    width = surf->w;
    height = surf->h;
    SDL_SetSurfaceRLE(surf, 1);
    
    // Initialize involution kernel
    conv_t conv_type;
    if(argc >= 3){
        if (strcmp(argv[2], "identity") == 0) conv_type = CONV_IDENTITY;
        else if (strcmp(argv[2], "edge") == 0) conv_type= CONV_EDGE;
        else if (strcmp(argv[2], "sharp") == 0) conv_type= CONV_SHARP;
        else if (strcmp(argv[2], "gauss") == 0) conv_type = CONV_GAUSS;
        else conv_type = CONV_IDENTITY;
    }
    switch(conv_type){
        case CONV_EDGE:
            filter[0] = -1; filter[1] = -1; filter[2] = -1; 
            filter[3] = -1; filter[4] = 8; filter[5] = -1; 
            filter[6] = -1; filter[7] = -1; filter[8] = -1; 
            break;
        case CONV_SHARP:
            filter[0] = 0; filter[1] = -1; filter[2] = 0; 
            filter[3] = -1; filter[4] = 5; filter[5] = -1; 
            filter[6] = 0; filter[7] = -1; filter[8] = 0; 
            break;
        case CONV_GAUSS:
            filter[0] = 1.0f/16; filter[1] = 1.0f/8; filter[2] = 1.0f/16; 
            filter[3] = 1.0f/8; filter[4] = 1.0f/4; filter[5] = 1.0f/8; 
            filter[6] = 1.0f/16; filter[7] = 1.0f/8; filter[8] = 1.0f/8; 
            break;
        default:
            filter[0] = 0; filter[1] = 0; filter[2] = 0; 
            filter[3] = 0; filter[4] = 1; filter[5] = 0; 
            filter[6] = 0; filter[7] = 0; filter[8] = 0; 
            break;
    }
    hipMemcpyToSymbolAsync(HIP_SYMBOL(filter_device), filter, sizeof(float)*9, 0, hipMemcpyHostToDevice);
    
    if(argc >= 4){
        if(strcmp(argv[3], "texture") == 0) withtex = true;
    }
    
    // Create window
	screen = SDL_CreateWindow("Original", 
                        100,
                        100,
                        width, height, SDL_WINDOW_SHOWN);
    if ( screen == NULL ) {
        fprintf(stderr, "Couldn't set up window: %s\n", SDL_GetError());
        exit(1);
    }
    screen_res = SDL_CreateWindow("Filtered", 
                        200,
                        100,
                        width, height, SDL_WINDOW_SHOWN);
    if ( screen_res == NULL ) {
        fprintf(stderr, "Couldn't set up window: %s\n", SDL_GetError());
        exit(1);
    }
    
    // Initialize CUDA
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    
    // Create renderer and texture
    ren = SDL_CreateRenderer(screen, -1, SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC);
    tex = SDL_CreateTextureFromSurface(ren, surf);
    
    // Show image
    SDL_RenderCopy(ren, tex, NULL, NULL);
    SDL_RenderPresent(ren);
    
    // Compute
    SDL_LockSurface(surf);
    float elapsed;
    if(withtex){
        elapsed = conv_texture_gpu((unsigned char*)surf->pixels, width, height);
    }else{
        elapsed = conv_global_gpu((unsigned char*)surf->pixels, width, height);
    }
    SDL_UnlockSurface(surf);
    
    // Show computed image
    ren_res = SDL_CreateRenderer(screen_res, -1, SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC);
    tex_res = SDL_CreateTextureFromSurface(ren_res, surf);
    SDL_RenderCopy(ren_res, tex_res, NULL, NULL);
    SDL_RenderPresent(ren_res);
    SDL_FreeSurface(surf);
    
    while (1) {
        SDL_WaitEvent(&event);
        if ((event.type == SDL_QUIT) || ((event.type == SDL_WINDOWEVENT) && (event.window.event == SDL_WINDOWEVENT_CLOSE))) break;
    }
    
    char s[100];
    sprintf(s, "Kernel execution time: %.4fms", elapsed);
    SDL_ShowSimpleMessageBox(SDL_MESSAGEBOX_INFORMATION, "Timing", s, screen);
    SDL_DestroyTexture(tex);
    SDL_DestroyRenderer(ren);
    SDL_DestroyWindow(screen);
    SDL_DestroyWindow(screen_res);
    exit(0);
}