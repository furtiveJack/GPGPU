#include <hip/hip_runtime.h>

#include <stdio.h>


class cuStopwatch{
    
    public:
        cuStopwatch();
        ~cuStopwatch();
        void start();
        float stop();

    private:
        float elapsedTime;
        bool started;
        hipEvent_t startTime;
        hipEvent_t endTime;
};

cuStopwatch::cuStopwatch(){
    started = false;
    elapsedTime = 0;
    hipError_t res = hipEventCreate(&startTime);
    if (res != 0)
        printf("Return code when recording startTime : %d\n", res);

    res = hipEventCreate(&endTime);
    if (res != 0)
        printf("Return code when recording endTime : %d\n", res);
}

cuStopwatch::~cuStopwatch(){
    hipEventDestroy(startTime);
    hipEventDestroy(endTime);
}

void cuStopwatch::start(){
    // todo: start the stopwatch, and ignore double start
    if (started) {
        return;
    }
    hipError_t res = hipEventRecord(startTime);
    if (res != 0)
        printf("Return code when recording startTime : %d\n", res);
    started = true;
}

float cuStopwatch::stop(){
    // todo: stop the stopwatch and return elapsed time, ignore invalid stops (e.g. stop when not yet started or double stop)
    if (! started) {
        return 0;
    }
    hipError_t res = hipEventRecord(endTime);
    if (res != 0)
        printf("Return code when recording endTime : %d\n", res);

    hipEventSynchronize(endTime);
    
    res = hipEventElapsedTime(&elapsedTime, startTime, endTime);
    if (res != 0)
        printf("Return code when computing elapsed time : %d\n", res);
    
    started = false;
    return elapsedTime;
}

